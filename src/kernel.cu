#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#define __HIPCC__
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char* msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3* dev_pos;
glm::vec3* dev_vel1;
glm::vec3* dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int* dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int* dev_particleGridIndices; // What grid cell is this particle in?

// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int* dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int* dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_coherent_pos;
glm::vec3* dev_coherent_vel2;
// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3* arr, float scale) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		glm::vec3 rand = generateRandomVec3(time, index);
		arr[index].x = scale * rand.x;
		arr[index].y = scale * rand.y;
		arr[index].z = scale * rand.z;
	}
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	// Don't forget to hipFree in  Boids::endSimulation.
	hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");
	// LOOK-1.2 - This is a typical CUDA kernel invocation.
	kernGenerateRandomPosArray << <fullBlocksPerGrid, blockSize >> > (1, numObjects,
		dev_pos, scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	// LOOK-2.1 computing grid params
	gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	// TODO-2.1 TODO-2.3 - Allocate additional buffers here.

	hipMallocAsync(&dev_particleArrayIndices, N * sizeof(int), hipStreamPerThread);
	checkCUDAErrorWithLine("hipMallocAsync dev_particleArrayIndices failed!");

	hipMallocAsync(&dev_particleGridIndices, N * sizeof(int), hipStreamPerThread);
	checkCUDAErrorWithLine("hipMallocAsync dev_particleGridIndices failed!");
	//Actually start and end indices only need gridCellCount * sizeof(int) bytes
	hipMallocAsync(&dev_gridCellStartIndices, gridCellCount * sizeof(int), hipStreamPerThread);
	checkCUDAErrorWithLine("hipMallocAsync dev_gridCellStartIndices failed!");

	hipMallocAsync(&dev_gridCellEndIndices, gridCellCount * sizeof(int), hipStreamPerThread);
	checkCUDAErrorWithLine("hipMallocAsync dev_gridCellEndIndices failed!");

	hipMallocAsync(&dev_coherent_pos, N * sizeof(glm::vec3), hipStreamPerThread);
	checkCUDAErrorWithLine("hipMallocAsync dev_coherent_pos failed!");

	hipMallocAsync(&dev_coherent_vel2, N * sizeof(glm::vec3), hipStreamPerThread);
	checkCUDAErrorWithLine("hipMallocAsync dev_coherent_vel2 failed!");


	hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float* vbodptr_positions, float* vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_vel1, vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
//This function calculates delta of velocity according to the rules
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	// Rule 2: boids try to stay a distance d away from each other
	// Rule 3: boids try to match the speed of surrounding boids
	  //x, y, z represents delta of velocity in 3 dimensions
	float neighbor_count = 0.f;
	float neighbor_count3 = 0.f;
	glm::vec3 center(0.0f, 0.0f, 0.0f);
	glm::vec3 seperate(0.0f, 0.0f, 0.0f);
	glm::vec3 cohesion(0.0f, 0.0f, 0.0f);
	glm::vec3 thisBoidp = pos[iSelf];
	glm::vec3 thisBoidv = vel[iSelf];
	glm::vec3 return_vel(0.0f, 0.0f, 0.0f);
	auto sqrt_distance = [](float x, float y, float z) {
		return __fsqrt_rn(x * x + y * y + z * z);
	};
	//iter all other boids
	for (int i = 0; i < N; i++)
	{
		if (i == iSelf) continue;
		float distance = sqrt_distance(thisBoidp.x - pos[i].x, thisBoidp.y - pos[i].y,
			thisBoidp.z - pos[i].z);
		if (distance < rule1Distance)
		{
			//Rule 1
			center.x += pos[i].x;
			center.y += pos[i].y;
			center.z += pos[i].z;
			neighbor_count += 1.0f;
		}
		if (distance < rule2Distance)
		{
			//Rule 2
			seperate.x -= pos[i].x - thisBoidp.x;
			seperate.y -= pos[i].y - thisBoidp.y;
			seperate.z -= pos[i].z - thisBoidp.z;
		}
		if (distance < rule3Distance)
		{
			//Rule 3
			cohesion.x += vel[i].x;
			cohesion.y += vel[i].y;
			cohesion.z += vel[i].z;
			neighbor_count3 += 1.0f;
		}
	}
	if (neighbor_count)
	{
		center.x /= neighbor_count;
		center.y /= neighbor_count;
		center.z /= neighbor_count;
		center.x = (center.x - thisBoidp.x) * rule1Scale;
		center.y = (center.y - thisBoidp.y) * rule1Scale;
		center.z = (center.z - thisBoidp.z) * rule1Scale;
	}
	seperate.x *= rule2Scale;
	seperate.y *= rule2Scale;
	seperate.z *= rule2Scale;
	if (neighbor_count3)
	{
		cohesion.x /= neighbor_count3;
		cohesion.y /= neighbor_count3;
		cohesion.z /= neighbor_count3;
		cohesion.x *= rule3Scale;
		cohesion.y *= rule3Scale;
		cohesion.z *= rule3Scale;
	}
	return_vel.x += center.x + seperate.x + cohesion.x;
	return_vel.y += center.y + seperate.y + cohesion.y;
	return_vel.z += center.z + seperate.z + cohesion.z;
	return return_vel;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
	glm::vec3* vel1, glm::vec3* vel2) {
	int iSelf = threadIdx.x + blockDim.x * blockIdx.x;
	if (iSelf >= N) return;
	// Compute a new velocity based on pos and vel1
		  //delta of velocity for each boid
	glm::vec3 return_vel;

	return_vel = vel1[iSelf] + computeVelocityChange(N, iSelf, pos, vel1);
	// Clamp the speed
	if (glm::length(return_vel) > maxSpeed)
	{
		return_vel = glm::normalize(return_vel) * maxSpeed;
	}
	// Record the new velocity into vel2. Question: why NOT vel1?
	 //Answer: ping-pong velocity
	vel2[iSelf] = return_vel;

}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3* pos, glm::vec3* vel) {
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

	thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	glm::vec3 gridMin, float inverseCellWidth,
	glm::vec3* pos, int* indices, int* gridIndices) {
	// TODO-2.1
	// - Label each boid with the index of its grid cell.
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= N) return;
	//get the coord of boid marked with grid cell
	//assume that pos.x is 0-9.999, divided by 10 then it goes to 0, pos.x is 0 in grid cell coord
	//but pos.x can be minus, so -gridMin makes it positive
	//so that we can get positive gridIndices
	glm::ivec3 boidPos = (pos[index] - gridMin) * inverseCellWidth;
	gridIndices[index] = gridIndex3Dto1D(boidPos.x, boidPos.y, boidPos.z, gridResolution);
	// - Set up a parallel array of integer indices as pointers to the actual
	indices[index] = index;
	//   boid data in pos and vel1/vel2

}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
	int* gridCellStartIndices, int* gridCellEndIndices) {
	// TODO-2.1
	// Identify the start point of each cell in the gridIndices array.
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= N) return;
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
	if (index == 0)
	{
		gridCellStartIndices[particleGridIndices[index]] = 0;
	}
	else if (index == N - 1)
	{
		gridCellEndIndices[particleGridIndices[index]] = N - 1;
	}
	else if (particleGridIndices[index] != particleGridIndices[index + 1])//2 boids in different grid cell
		//judge on the boundary wont cause mem write confliction
	{
		gridCellEndIndices[particleGridIndices[index]] = index;//this cell ends at index
		gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;//another cell starts at index+1
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int* gridCellStartIndices, int* gridCellEndIndices,
	int* particleArrayIndices,
	glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
	// TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= N) return;
	glm::vec3 center(0.0f, 0.0f, 0.0f);
	glm::vec3 seperate(0.0f, 0.0f, 0.0f);
	glm::vec3 velocity(0.0f, 0.0f, 0.0f);
	glm::vec3 return_vel(0.0f, 0.0f, 0.0f);
	glm::vec3 thisBoid = pos[index];
	int neighbor_count = 0;
	int neighbor_count3 = 0;
	float distance = 0;
	// - Identify the grid cell that this particle is in
	glm::ivec3 boidPos = (thisBoid - gridMin) * inverseCellWidth;
	int x = boidPos.x;
	int y = boidPos.y;
	int z = boidPos.z;
	// - Identify which cells may contain neighbors. This isn't always 8.
	//so calculate 3*3*3 neighbors, these neighbor's grid cell index can
	//be calculated by gridIndex3Dto1D
	for (int k = -1; k <= 1; k++)
	{
		for (int j = -1; j <= 1; j++)
		{
			for (int i = -1; i <= 1; i++)
			{
				int near_x = x + i;
				int near_y = y + j;
				int near_z = z + k;
				//boundary check, near_?  = [0, gridResolution)
				near_x = imax(near_x, 0);
				near_y = imax(near_y, 0);
				near_z = imax(near_z, 0);
				near_x = imin(near_x, gridResolution - 1);
				near_y = imin(near_y, gridResolution - 1);
				near_z = imin(near_z, gridResolution - 1);
				//near_? to grid cell index
				int nearGridCellIndex = gridIndex3Dto1D(near_x, near_y, near_z, gridResolution);
				// - For each cell, read the start/end indices in the boid pointer array.
				if (gridCellStartIndices[nearGridCellIndex] != -1)//is not empty
				{
					for (int indices = gridCellStartIndices[nearGridCellIndex]; indices <= gridCellEndIndices[nearGridCellIndex]; indices++)
					{
						// - Access each boid in the cell and compute velocity change from
						//   the boids rules, if this boid is within the neighborhood distance.
						int bindex = particleArrayIndices[indices];
						if (bindex != index)
						{
							distance = glm::distance(pos[bindex], thisBoid);
							//Rule1
							if (distance < rule1Distance)
							{
								center += pos[bindex];
								neighbor_count++;
							}
							//Rule2
							if (distance < rule2Distance)
							{
								seperate -= (pos[bindex] - thisBoid);
							}
							//Rule3
							if (distance < rule3Distance)
							{
								velocity += vel1[bindex];
								neighbor_count3++;
							}
						}
					}
				}
			}
		}
	}
	if (neighbor_count)
	{
		center /= neighbor_count;
		center = (center - thisBoid) * rule1Scale;
	}
	if (neighbor_count3)
	{
		velocity /= neighbor_count3;
		velocity *= rule3Scale;
	}
	seperate *= rule2Scale;
	// - Clamp the speed change before putting the new speed in vel2
	return_vel = vel1[index] + center + seperate + velocity;
	if (glm::length(return_vel) > maxSpeed)
	{
		return_vel = glm::normalize(return_vel) * maxSpeed;
	}
	vel2[index] = return_vel;
}
/// <summary>
/// The coherent manner sorts the pos and vel1, so that the memory is
/// continuous, which makes cuda run faster.
/// </summary>
/// <param name="N"></param>
/// <param name="gridResolution"></param>
/// <param name="gridMin"></param>
/// <param name="inverseCellWidth"></param>
/// <param name="cellWidth"></param>
/// <param name="gridCellStartIndices"></param>
/// <param name="gridCellEndIndices"></param>
/// <param name="pos">dev_coherent_pos</param>
/// <param name="vel1">dev_coherent_vel2</param>
/// <param name="vel2"></param>
/// <returns></returns>
__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int* gridCellStartIndices, int* gridCellEndIndices,
	glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
	// TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= N) return;
	glm::vec3 center(0.0f, 0.0f, 0.0f);
	glm::vec3 seperate(0.0f, 0.0f, 0.0f);
	glm::vec3 velocity(0.0f, 0.0f, 0.0f);
	glm::vec3 return_vel(0.0f, 0.0f, 0.0f);
	glm::vec3 thisBoid = pos[index];
	int neighbor_count = 0;
	int neighbor_count3 = 0;
	float distance = 0;
	// - Identify the grid cell that this particle is in
	glm::ivec3 boidPos = (thisBoid - gridMin) * inverseCellWidth;
	int x = boidPos.x;
	int y = boidPos.y;
	int z = boidPos.z;
	// - Identify which cells may contain neighbors. This isn't always 8.
	//so calculate 3*3*3 neighbors, these neighbor's grid cell index can
	//be calculated by gridIndex3Dto1D
	//   DIFFERENCE: For best results, consider what order the cells should be
	//   checked in to maximize the memory benefits of reordering the boids data.
	//  Actually no difference, I used to range them as z, y, x, benefit max
	for (int k = -1; k <= 1; k++)
	{
		for (int j = -1; j <= 1; j++)
		{
			for (int i = -1; i <= 1; i++)
			{
				int near_x = x + i;
				int near_y = y + j;
				int near_z = z + k;
				//boundary check, near_?  = [0, gridResolution)
				near_x = imax(near_x, 0);
				near_y = imax(near_y, 0);
				near_z = imax(near_z, 0);
				near_x = imin(near_x, gridResolution - 1);
				near_y = imin(near_y, gridResolution - 1);
				near_z = imin(near_z, gridResolution - 1);
				//near_? to grid cell index
				int nearGridCellIndex = gridIndex3Dto1D(near_x, near_y, near_z, gridResolution);
				// - For each cell, read the start/end indices in the boid pointer array.
				if (gridCellStartIndices[nearGridCellIndex] != -1)//is not empty
				{
					for (int indices = gridCellStartIndices[nearGridCellIndex]; indices <= gridCellEndIndices[nearGridCellIndex]; indices++)
					{
						// - Access each boid in the cell and compute velocity change from
						//   the boids rules, if this boid is within the neighborhood distance.
						int bindex = indices;
						if (bindex != index)
						{
							distance = glm::distance(pos[bindex], thisBoid);
							//Rule1
							if (distance < rule1Distance)
							{
								center += pos[bindex];
								neighbor_count++;
							}
							//Rule2
							if (distance < rule2Distance)
							{
								seperate -= (pos[bindex] - thisBoid);
							}
							//Rule3
							if (distance < rule3Distance)
							{
								velocity += vel1[bindex];
								neighbor_count3++;
							}
						}
					}
				}
			}
		}
	}
	if (neighbor_count)
	{
		center /= neighbor_count;
		center = (center - thisBoid) * rule1Scale;
	}
	if (neighbor_count3)
	{
		velocity /= neighbor_count3;
		velocity *= rule3Scale;
	}
	seperate *= rule2Scale;
	// - Clamp the speed change before putting the new speed in vel2
	return_vel = vel1[index] + center + seperate + velocity;
	if (glm::length(return_vel) > maxSpeed)
	{
		return_vel = glm::normalize(return_vel) * maxSpeed;
	}
	vel2[index] = return_vel;

}

__global__	 void kernCoherentPosVel(int N, int* dev_particleArrayIndices, 
	glm::vec3* pos, glm::vec3* vel,
	glm::vec3* coherentpos, glm::vec3* coherentvel)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= N) return;
	//coherentindex is where the value truely locates in pos and vel
	int coherentindex = dev_particleArrayIndices[index];
	//map them
	coherentpos[index] = pos[coherentindex];
	coherentvel[index] = vel[coherentindex];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	// TODO-1.2 ping-pong the velocity buffers
	dim3 grids((numObjects - 1) / blockSize + 1);

	kernUpdateVelocityBruteForce << <grids, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
	kernUpdatePos << <grids, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernel update failed!");
	//Err.. can this be ping-pong?
	//It can! Another way is std::swap()
	hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

}

void Boids::stepSimulationScatteredGrid(float dt) {
	// TODO-2.1
	dim3 grids((numObjects - 1) / blockSize + 1);
	dim3 gridsGridCell((gridCellCount - 1) / blockSize + 1);
	//reset start and end indices, in case of empty grid cell
	kernResetIntBuffer << <gridsGridCell, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <gridsGridCell, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
	// Uniform Grid Neighbor search using Thrust sort.
	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	kernComputeIndices << <grids, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::device_ptr<int>dev_thrust_arrayIndice(dev_particleArrayIndices);
	thrust::device_ptr<int>dev_thrust_gridIndice(dev_particleGridIndices);
	thrust::sort_by_key(dev_thrust_gridIndice, dev_thrust_gridIndice + numObjects, dev_thrust_arrayIndice);
	checkCUDAErrorWithLine("thrust sort_by_key failed!");
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd << <grids, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered << <grids, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	// - Update positions
	kernUpdatePos << <grids, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
	// - Ping-pong buffers as needed
	hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

}

void Boids::stepSimulationCoherentGrid(float dt) {
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	dim3 grids((numObjects - 1) / blockSize + 1);
	dim3 gridsGridCell((gridCellCount - 1) / blockSize + 1);
	//reset start and end indices, in case of empty grid cell
	kernResetIntBuffer << <gridsGridCell, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <gridsGridCell, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	// In Parallel:
	// - Label each particle with its array index as well as its grid index.
	//   Use 2x width grids
	kernComputeIndices << <grids, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::device_ptr<int>dev_thrust_arrayIndice(dev_particleArrayIndices);
	thrust::device_ptr<int>dev_thrust_gridIndice(dev_particleGridIndices);
	thrust::sort_by_key(dev_thrust_gridIndice, dev_thrust_gridIndice + numObjects, dev_thrust_arrayIndice);
	checkCUDAErrorWithLine("thrust sort_by_key failed!");
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd << <grids, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	kernCoherentPosVel<<<grids, blockSize>>>(numObjects, dev_particleArrayIndices, dev_pos, dev_vel1, dev_coherent_pos, dev_coherent_vel2);
	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent << <grids, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices, dev_coherent_pos, dev_coherent_vel2, dev_vel1);
	// - Update positions
	kernUpdatePos<<<grids, blockSize>>>(numObjects, dt, dev_coherent_pos, dev_vel1);
	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	//hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
	hipMemcpy(dev_pos, dev_coherent_pos, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::endSimulation() {
	hipFree(dev_vel1);
	hipFree(dev_vel2);
	hipFree(dev_pos);

	// TODO-2.1 TODO-2.3 - Free any additional buffers here.
	hipFreeAsync(dev_particleArrayIndices, hipStreamPerThread);
	hipFreeAsync(dev_particleGridIndices, hipStreamPerThread);
	hipFreeAsync(dev_gridCellStartIndices, hipStreamPerThread);
	hipFreeAsync(dev_gridCellEndIndices, hipStreamPerThread);
	hipFreeAsync(dev_coherent_pos, hipStreamPerThread);
	hipFreeAsync(dev_coherent_vel2, hipStreamPerThread);
}

void Boids::unitTest() {
	// LOOK-1.2 Feel free to write additional tests here.

	// test unstable sort
	int* dev_intKeys;
	int* dev_intValues;
	int N = 10;

	std::unique_ptr<int[]>intKeys{ new int[N] };
	std::unique_ptr<int[]>intValues{ new int[N] };

	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;

	hipMalloc((void**)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

	hipMalloc((void**)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// How to copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// cleanup
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}
